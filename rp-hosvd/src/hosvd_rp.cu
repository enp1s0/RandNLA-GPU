#include "hip/hip_runtime.h"
#include <cutf/memory.hpp>
#include <cutf/cutensor.hpp>
#include <cuta/cutensor_utils.hpp>
#include <cuda_common.hpp>
#include "hosvd_test.hpp"

#ifdef TIME_BREAKDOWN
#define CUTF_PROFILE_START_TIMER(name) CUTF_CHECK_ERROR(hipStreamSynchronize(hip_stream));profiler.start_timer_sync(name)
#define CUTF_PROFILE_STOP_TIMER(name)  CUTF_CHECK_ERROR(hipStreamSynchronize(hip_stream));profiler.stop_timer_sync(name)
#else
#define CUTF_PROFILE_START_TIMER(name)
#define CUTF_PROFILE_STOP_TIMER(name)
#endif

void mtk::rsvd_test::hosvd_rp::prepare() {
	working_memory.alloc_ptr = nullptr;
	contraction_working_mem_ptr = nullptr;

	const std::size_t prod = cuta::utils::get_num_elements(input_tensor_mode);
	std::size_t max_rand_matrix_m = 0;
	std::size_t max_rand_matrix_n = 0;
	std::size_t max_rand_matrix_size = 0;
	std::size_t max_rand_matrix_index = 0;
	for (unsigned i = 0; i < input_tensor_mode.size(); i++) {
		const auto rand_m = prod / input_tensor_mode[i].second;
		const auto rand_n = core_tensor_mode[i].second;

		if (rand_m * rand_n > max_rand_matrix_size) {
			max_rand_matrix_index = i;
			max_rand_matrix_m = rand_m;
			max_rand_matrix_n = rand_n;
			max_rand_matrix_size = rand_m * rand_n;
		}
	}

	random_projection.set_config(
			input_tensor_mode[max_rand_matrix_index].second, max_rand_matrix_m, max_rand_matrix_n,
			hip_stream
			);
	random_projection.allocate_working_memory();

	// For TTGT
	working_memory.ttgt_size = prod;

	// QR
	Q_tensor_mode.resize                 (input_tensor_mode.size());
	Q_tensor_desc.resize                 (input_tensor_mode.size());
	Q_tensor_alignment_requirement.resize(input_tensor_mode.size());
	working_memory.tau_size = 0;
	working_memory.qr_size  = 0;
	for (unsigned i = 0; i < input_tensor_mode.size(); i++) {
		Q_tensor_mode[i].push_back(input_tensor_mode[i]);
		Q_tensor_mode[i].push_back(core_tensor_mode[i]);
		Q_tensor_desc[i] = cuta::cutensor::get_descriptor<float>(cutensor_handle, Q_tensor_mode[i]);
		CUTT_CHECK_ERROR(cutensorGetAlignmentRequirement(&cutensor_handle, Q_ptr[i], &Q_tensor_desc[i], &Q_tensor_alignment_requirement[i]));

		int qr_size_0, qr_size_1;
		CUTF_CHECK_ERROR(cutf::cusolver::dn::geqrf_buffer_size(
					cusolver_handle,
					input_tensor_mode[i].second, core_tensor_mode[i].second,
					Q_ptr[i], input_tensor_mode[i].second,
					&qr_size_0
					));

		CUTF_CHECK_ERROR(hipsolverDnSorgqr_bufferSize(
					cusolver_handle,
					input_tensor_mode[i].second, core_tensor_mode[i].second,
					core_tensor_mode[i].second,
					Q_ptr[i], input_tensor_mode[i].second,
					working_memory.tau_ptr,
					&qr_size_1
					));
		working_memory.geqrf_size.push_back(qr_size_0);
		working_memory.orgqr_size.push_back(qr_size_1);
		working_memory.tau_size = std::max<std::size_t>(working_memory.tau_size, core_tensor_mode[i].second);
		working_memory.qr_size  = std::max<std::size_t>(working_memory.qr_size , std::max(qr_size_0, qr_size_1)) + 1;
	}

	// Allocation (1)
	working_memory.alloc_size = working_memory.ttgt_size + working_memory.qr_size + working_memory.tau_size + 1 /*dev*/;

	working_memory.alloc_ptr = cutf::memory::malloc_async<float>(working_memory.alloc_size, hip_stream);

	working_memory.ttgt_ptr = working_memory.alloc_ptr;
	working_memory.qr_ptr   = working_memory.ttgt_ptr + working_memory.ttgt_size;
	working_memory.tau_ptr  = working_memory.qr_ptr + working_memory.qr_size;
	working_memory.dev_ptr  = reinterpret_cast<int*>(working_memory.tau_ptr) + working_memory.tau_size;

	// Resize
	contraction_desc.resize            (input_tensor_mode.size());
	contraction_working_mem_size.resize(input_tensor_mode.size());
	contraction_find.resize            (input_tensor_mode.size());
	contraction_plan.resize            (input_tensor_mode.size());
	// Tensor contraction
	tmp_core_tensor_mode.resize                 (input_tensor_mode.size() + 1);
	tmp_core_tensor_desc.resize                 (input_tensor_mode.size() + 1);
	tmp_core_tensor_alignment_requirement.resize(input_tensor_mode.size() + 1);
	tmp_core_tensor_mode[0] = input_tensor_mode;
	tmp_core_tensor_desc[0] = cuta::cutensor::get_descriptor<float>(cutensor_handle, input_tensor_mode);
	CUTT_CHECK_ERROR(cutensorGetAlignmentRequirement(&cutensor_handle, A_ptr, &tmp_core_tensor_desc[0], &tmp_core_tensor_alignment_requirement[0]));
	for (unsigned i = 1; i <= input_tensor_mode.size(); i++) {
		auto t_mode = tmp_core_tensor_mode[i - 1];
		t_mode[i - 1] = core_tensor_mode[i - 1];
		tmp_core_tensor_mode[i] = t_mode;

		float* mem_ptr;
		if (i == input_tensor_mode.size()) {
			mem_ptr = S_ptr;
		} else if (i % 2 == 0) {
			mem_ptr = A_ptr;
		} else {
			mem_ptr = working_memory.alloc_ptr;
		}
		tmp_core_tensor_desc[i] = cuta::cutensor::get_descriptor<float>(cutensor_handle, tmp_core_tensor_mode[i]);
		CUTT_CHECK_ERROR(cutensorGetAlignmentRequirement(&cutensor_handle, mem_ptr, &tmp_core_tensor_desc[i], &tmp_core_tensor_alignment_requirement[i]));

		// Set contraction descriptor
		CUTT_CHECK_ERROR(cutensorInitContractionDescriptor(&cutensor_handle, &contraction_desc[i - 1],
				&tmp_core_tensor_desc[i - 1], cuta::cutensor::get_extent_list_in_int(tmp_core_tensor_mode[i - 1]).data(), tmp_core_tensor_alignment_requirement[i - 1],
				&Q_tensor_desc       [i - 1], cuta::cutensor::get_extent_list_in_int(Q_tensor_mode       [i - 1]).data(), Q_tensor_alignment_requirement       [i - 1],
				&tmp_core_tensor_desc[i    ], cuta::cutensor::get_extent_list_in_int(tmp_core_tensor_mode[i    ]).data(), tmp_core_tensor_alignment_requirement[i    ],
				&tmp_core_tensor_desc[i    ], cuta::cutensor::get_extent_list_in_int(tmp_core_tensor_mode[i    ]).data(), tmp_core_tensor_alignment_requirement[i    ],
				cuta::cutensor::get_compute_type<float>()));

		// Set find
		CUTT_CHECK_ERROR(cutensorInitContractionFind(&cutensor_handle, &contraction_find[i - 1], HIPTENSOR_ALGO_DEFAULT));

		// calculate working memory size
		CUTT_CHECK_ERROR(cutensorContractionGetWorkspace(&cutensor_handle, &contraction_desc[i - 1], &contraction_find[i - 1], CUTENSOR_WORKSPACE_RECOMMENDED, &contraction_working_mem_size[i - 1]));

		// set plan
		CUTT_CHECK_ERROR(cutensorInitContractionPlan(&cutensor_handle, &contraction_plan[i - 1], &contraction_desc[i - 1], &contraction_find[i - 1], contraction_working_mem_size[i - 1]));
	}
	// Calc working memory size
	contraction_working_mem_size_max = 0;
	for (unsigned i = 0; i < input_tensor_mode.size(); i++) {
		contraction_working_mem_size_max = std::max(contraction_working_mem_size_max, contraction_working_mem_size[i]);
	}
	contraction_working_mem_ptr = cutf::memory::malloc_async<uint8_t>(contraction_working_mem_size_max, hip_stream);

	random_projection.gen_rand(100);
}

void mtk::rsvd_test::hosvd_rp::clean() {
	cutf::memory::free_async(working_memory.alloc_ptr, hip_stream);
	working_memory.alloc_ptr = nullptr;
	cutf::memory::free_async(contraction_working_mem_ptr, hip_stream);
	contraction_working_mem_ptr = nullptr;
	random_projection.free_working_memory();
}

void mtk::rsvd_test::hosvd_rp::run() {
	const float alpha = 1.f;
	const float beta = 0.f;
	// Transpose the tensor
	for (unsigned i = 0; i < input_tensor_mode.size(); i++) {
		// Transpose
		std::vector<std::string> reshaped_mode_order(input_tensor_mode.size());
		const auto target_mode_name = input_tensor_mode[i].first;
		reshaped_mode_order[0] = target_mode_name;
		for (unsigned j = 0, k = 1; j < input_tensor_mode.size(); j++) {
			if (i != j) {
				reshaped_mode_order[k++] = input_tensor_mode[j].first;
			}
		}
		const auto permutated_mode = cuta::utils::get_permutated_mode(input_tensor_mode, reshaped_mode_order);
		const auto desc_A = cuta::cutensor::get_descriptor<float>(cutensor_handle, input_tensor_mode);
		const auto desc_B = cuta::cutensor::get_descriptor<float>(cutensor_handle, permutated_mode);
		CUTF_PROFILE_START_TIMER("reshape");
		//cuttExecute(cutt_handle_list[i], A_ptr, working_memory.ttgt_ptr);
		CUTF_CHECK_ERROR(hiptensorPermutation(
					&cutensor_handle,
					&alpha,
					A_ptr,
					&desc_A,
					cuta::cutensor::get_extent_list_in_int(input_tensor_mode).data(),
					working_memory.ttgt_ptr,
					&desc_B,
					cuta::cutensor::get_extent_list_in_int(permutated_mode).data(),
					cuta::cutensor::get_data_type<float>(),
					hip_stream
					));
		CUTF_PROFILE_STOP_TIMER("reshape");
		CUTF_PROFILE_START_TIMER("random_projection");
		// Rand projection
		random_projection.apply(
				input_tensor_mode[i].second, cuta::utils::get_num_elements(input_tensor_mode) / input_tensor_mode[i].second, core_tensor_mode[i].second,
				Q_ptr[i], input_tensor_mode[i].second,
				working_memory.ttgt_ptr, input_tensor_mode[i].second
				);
		CUTF_PROFILE_STOP_TIMER("random_projection");
		CUTF_PROFILE_START_TIMER("qr");
		// QR
		CUTF_CHECK_ERROR(cutf::cusolver::dn::geqrf(
					cusolver_handle,
					input_tensor_mode[i].second, core_tensor_mode[i].second,
					Q_ptr[i], input_tensor_mode[i].second,
					working_memory.tau_ptr,
					working_memory.qr_ptr,
					working_memory.geqrf_size[i],
					working_memory.dev_ptr
					));
		CUTF_CHECK_ERROR(hipsolverDnSorgqr(
					cusolver_handle,
					input_tensor_mode[i].second, core_tensor_mode[i].second,
					core_tensor_mode[i].second,
					Q_ptr[i], input_tensor_mode[i].second,
					working_memory.tau_ptr,
					working_memory.qr_ptr,
					working_memory.orgqr_size[i],
					working_memory.dev_ptr
					));
		CUTF_PROFILE_STOP_TIMER("qr");
	}
	// Compute the core tensor
	float *input_ptr;
	float *output_ptr;
	for (unsigned i = 0; i < input_tensor_mode.size(); i++) {
		if (i % 2 == 0) {
			input_ptr = A_ptr;
		} else {
			input_ptr = working_memory.alloc_ptr;
		}
		if (i == input_tensor_mode.size() - 1) {
			output_ptr = S_ptr;
		} else if (i % 2 == 0) {
			output_ptr = working_memory.alloc_ptr;
		} else {
			output_ptr = A_ptr;
		}
		CUTF_PROFILE_START_TIMER("tensor_contraction");
		CUTT_CHECK_ERROR(hiptensorContraction(&cutensor_handle,
				&contraction_plan[i],
				reinterpret_cast<const void*>(&alpha), input_ptr, Q_ptr[i],
				reinterpret_cast<const void*>(&beta), output_ptr, output_ptr,
				contraction_working_mem_ptr, contraction_working_mem_size[i], 0
				));
		CUTF_PROFILE_STOP_TIMER("tensor_contraction");
	}
}
