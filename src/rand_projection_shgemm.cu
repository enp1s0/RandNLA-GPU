#include <cutf/memory.hpp>
#include <curand_fp16/curand_fp16.hpp>
#include <rand_projection_base.hpp>

void mtk::rsvd_test::random_projection_shgemm::gen_rand(const std::uint64_t seed) {
	mtk::curand_fp16::generator_t gen;
	mtk::curand_fp16::create(gen, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
	mtk::curand_fp16::set_seed(gen, seed);
	mtk::curand_fp16::set_cuda_stream(gen, hip_stream);
	mtk::curand_fp16::uniform(gen, rand_matrix_ptr, get_src_n() * get_target_rank());
	mtk::curand_fp16::destroy(gen);
}

void mtk::rsvd_test::random_projection_shgemm::apply(
		float* const dst_ptr, const std::size_t ldd,
		float* const src_ptr, const std::size_t lds
		) {
	const float alpha = 1.0f, beta = 0.0f;
	mtk::shgemm::shgemm(
				shgemm_handle,
				mtk::shgemm::op_n, mtk::shgemm::op_t,
				get_src_m(), get_target_rank(), get_src_n(),
				&alpha,
				src_ptr, lds,
				rand_matrix_ptr, get_target_rank(),
				&beta,
				dst_ptr, ldd
				);
}

void mtk::rsvd_test::random_projection_shgemm::allocate_working_memory() {
	rand_matrix_ptr = cutf::memory::malloc_async<half>(get_src_n() * get_target_rank(), hip_stream);
}

void mtk::rsvd_test::random_projection_shgemm::free_working_memory() {
	cutf::memory::free_async(rand_matrix_ptr, hip_stream);
}
