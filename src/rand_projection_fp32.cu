#include <cutf/hiprand.hpp>
#include <cutf/cublas.hpp>
#include <cutf/memory.hpp>
#include <rand_projection_base.hpp>

void mtk::rsvd_test::random_projection_fp32::gen_rand(const std::uint64_t seed) {
	auto cugen = cutf::hiprand::get_curand_unique_ptr(HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
	CUTF_CHECK_ERROR(hiprandSetPseudoRandomGeneratorSeed(*cugen.get(), seed));
	CUTF_CHECK_ERROR(hiprandSetStream(*cugen.get(), hip_stream));
	CUTF_CHECK_ERROR(cutf::hiprand::generate_uniform(*cugen.get(), rand_matrix_ptr, get_src_n() * get_target_rank()));
}

void mtk::rsvd_test::random_projection_fp32::apply(
		float* const dst_ptr, const std::size_t ldd,
		float* const src_ptr, const std::size_t lds
		) {
	const float alpha = 1.0f, beta = 0.0f;
	CUTF_CHECK_ERROR(cutf::cublas::gemm(
				cublas_handle,
				HIPBLAS_OP_N, HIPBLAS_OP_T,
				get_src_m(), get_target_rank(), get_src_n(),
				&alpha,
				src_ptr, lds,
				rand_matrix_ptr, get_target_rank(),
				&beta,
				dst_ptr, ldd
				));
}

void mtk::rsvd_test::random_projection_fp32::allocate_working_memory() {
	rand_matrix_ptr = cutf::memory::malloc_async<float>(get_src_n() * get_target_rank(), hip_stream);
}

void mtk::rsvd_test::random_projection_fp32::free_working_memory() {
	cutf::memory::free_async(rand_matrix_ptr, hip_stream);
}
