#include "hip/hip_runtime.h"
#include <iostream>
#include <rsvd_test.hpp>
#include <cutf/memory.hpp>
#include <cutf/cusolver.hpp>
#include <cutf/stream.hpp>

constexpr unsigned max_log_m = 13;
constexpr unsigned max_log_n = 13;
constexpr unsigned n_svdj_iter = 10;

namespace {
void evaluate(
		const std::string test_name,
		mtk::rsvd_test::rsvd_base& rsvd,
		const unsigned n_tests,
		hipStream_t hip_stream
		) {
	std::printf("%s,%u,%u,%u,%u,%u,",
			test_name.c_str(),
			rsvd.get_m(),
			rsvd.get_n(),
			rsvd.get_k(),
			rsvd.get_p(),
			rsvd.get_n_svdj_iter()
			);
	std::printf("%u\n", n_tests);
	const auto A_size = rsvd.get_m() * rsvd.get_n();
	const auto S_size = std::min(rsvd.get_m(), rsvd.get_n());
	const auto U_size = rsvd.get_m() * (rsvd.get_k() + rsvd.get_p());
	const auto V_size = rsvd.get_n() * (rsvd.get_k() + rsvd.get_p());

	auto A_ptr = cutf::memory::malloc_async<float>(A_size, hip_stream);
	auto U_ptr = cutf::memory::malloc_async<float>(U_size, hip_stream);
	auto S_ptr = cutf::memory::malloc_async<float>(S_size, hip_stream);
	auto V_ptr = cutf::memory::malloc_async<float>(V_size, hip_stream);

	rsvd.set_input_ptr(A_ptr);
	rsvd.set_output_ptr(U_ptr, S_ptr, V_ptr);

	try {
		rsvd.prepare();
		rsvd.run();
		rsvd.clean();
	} catch (const std::exception& e) {
		std::printf("%s\n", e.what());
	}

	cutf::memory::free_async<float>(A_ptr, hip_stream);
	cutf::memory::free_async<float>(U_ptr, hip_stream);
	cutf::memory::free_async<float>(S_ptr, hip_stream);
	cutf::memory::free_async<float>(V_ptr, hip_stream);
}
} // noname namespace

int main() {
	auto hip_stream  = cutf::stream::get_stream_unique_ptr();
	auto cusolver_handle = cutf::cusolver::dn::get_handle_unique_ptr();
	auto cusolver_params = cutf::cusolver::dn::get_params_unique_ptr();
	CUTF_CHECK_ERROR(hipsolverSetStream(*cusolver_handle.get(), *hip_stream.get()));
	CUTF_CHECK_ERROR(hipsolverDnSetAdvOptions(*cusolver_params.get(), HIPSOLVERDN_GETRF, HIPSOLVER_ALG_0));

	for (unsigned log_m = 5; log_m <= max_log_m; log_m++) {
		for (unsigned log_n = 5; log_n <= max_log_n; log_n++) {
			const auto max_log_k = std::min(log_m, log_n);
			for (unsigned log_k = 4; log_k <= max_log_k - 1; log_k++) {
				const auto m = 1u << log_m;
				const auto n = 1u << log_n;
				const auto k = 1u << log_k;
				const auto p = k;
				if (k + p > std::min(m, n)) {
					break;
				}

				mtk::rsvd_test::rsvd_cusolver rsvd_cusolver(
						*cusolver_handle.get(),
						*cusolver_params.get(),
						m, n, k, p, n_svdj_iter,
						nullptr, m,
						nullptr, m,
						nullptr,
						nullptr, n,
						*hip_stream.get()
						);

				evaluate("cusolver", rsvd_cusolver, 10, *hip_stream.get());
			}
		}
	}
}
