#include "hip/hip_runtime.h"
#include <iostream>
#include <rsvd_test.hpp>
#include <cutf/memory.hpp>
#include <cutf/cusolver.hpp>
#include <cutf/stream.hpp>
#include <cutf/hiprand.hpp>

constexpr unsigned max_log_m = 13;
constexpr unsigned max_log_n = 13;
constexpr unsigned n_svdj_iter = 10;

namespace {
void evaluate(
		const std::string test_name,
		mtk::rsvd_test::rsvd_base& rsvd,
		const unsigned n_tests,
		hipStream_t const hip_stream
		) {
	std::printf("%s,%u,%u,%u,%u,%u,",
			test_name.c_str(),
			rsvd.get_m(),
			rsvd.get_n(),
			rsvd.get_k(),
			rsvd.get_p(),
			rsvd.get_n_svdj_iter()
			);
	const auto A_size = rsvd.get_m() * rsvd.get_n();
	const auto S_size = std::min(rsvd.get_m(), rsvd.get_n());
	const auto U_size = rsvd.get_m() * (rsvd.get_k() + rsvd.get_p());
	const auto V_size = rsvd.get_n() * (rsvd.get_k() + rsvd.get_p());

	auto A_ptr = cutf::memory::malloc_async<float>(A_size, hip_stream);
	auto U_ptr = cutf::memory::malloc_async<float>(U_size, hip_stream);
	auto S_ptr = cutf::memory::malloc_async<float>(S_size, hip_stream);
	auto V_ptr = cutf::memory::malloc_async<float>(V_size, hip_stream);

	rsvd.set_input_ptr(A_ptr);
	rsvd.set_output_ptr(U_ptr, S_ptr, V_ptr);

	// Initialize the input matrix
	const uint64_t seed = 10;
	auto cugen = cutf::hiprand::get_curand_unique_ptr(HIPRAND_RNG_PSEUDO_MT19937);
	CUTF_CHECK_ERROR(hiprandSetPseudoRandomGeneratorSeed(*cugen.get(), seed));
	rsvd.prepare();

	auto elapsed_time_sum = 0.;
	for (unsigned i = 0; i < n_tests; i++) {
		CUTF_CHECK_ERROR(cutf::hiprand::generate_uniform(*cugen.get(), A_ptr, A_size));

		try {
			hipStreamSynchronize(hip_stream);
			const auto start_clock = std::chrono::system_clock::now();
			rsvd.run();
			hipStreamSynchronize(hip_stream);
			const auto end_clock = std::chrono::system_clock::now();
			const auto elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>(end_clock - start_clock).count() * 1e-6;
			elapsed_time_sum += elapsed_time;
			// Calculate the residual and max relative error

		} catch (const std::exception& e) {
			std::printf("%s\n", e.what());
		}
	}
	std::printf("%e,", elapsed_time_sum / n_tests);

	rsvd.clean();
	cutf::memory::free_async<float>(A_ptr, hip_stream);
	cutf::memory::free_async<float>(U_ptr, hip_stream);
	cutf::memory::free_async<float>(S_ptr, hip_stream);
	cutf::memory::free_async<float>(V_ptr, hip_stream);
	std::printf("%u\n", n_tests);
}
} // noname namespace

int main() {
	auto hip_stream  = cutf::stream::get_stream_unique_ptr();
	auto cusolver_handle = cutf::cusolver::dn::get_handle_unique_ptr();
	auto cusolver_params = cutf::cusolver::dn::get_params_unique_ptr();
	CUTF_CHECK_ERROR(hipsolverSetStream(*cusolver_handle.get(), *hip_stream.get()));
	CUTF_CHECK_ERROR(hipsolverDnSetAdvOptions(*cusolver_params.get(), HIPSOLVERDN_GETRF, HIPSOLVER_ALG_0));

	for (unsigned log_m = 5; log_m <= max_log_m; log_m++) {
		for (unsigned log_n = 5; log_n <= max_log_n; log_n++) {
			const auto max_log_k = std::min(log_m, log_n);
			for (unsigned log_k = 4; log_k <= max_log_k - 1; log_k++) {
				const auto m = 1u << log_m;
				const auto n = 1u << log_n;
				const auto k = 1u << log_k;
				const auto p = k;
				if (k + p > std::min(m, n)) {
					break;
				}

				mtk::rsvd_test::rsvd_cusolver rsvd_cusolver(
						*cusolver_handle.get(),
						*cusolver_params.get(),
						m, n, k, p, n_svdj_iter,
						nullptr, m,
						nullptr, m,
						nullptr,
						nullptr, n,
						*hip_stream.get()
						);

				evaluate("cusolver", rsvd_cusolver, 10, *hip_stream.get());
			}
		}
	}
}
