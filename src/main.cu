#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <vector>
#include <rsvd_test.hpp>
#include <input_matrix.hpp>
#include <cutf/memory.hpp>
#include <cutf/cusolver.hpp>
#include <cutf/stream.hpp>
#include <cutf/hiprand.hpp>
#include <mateval/comparison_cuda.hpp>

constexpr unsigned max_log_m = 10;
constexpr unsigned max_log_n = 10;
constexpr unsigned n_svdj_iter = 10;

namespace {
void print_csv_header() {
	std::printf("implementation,matrix,m,n,k,p,n_svdj_iter,residual,u_orthogonality,v_orthogonality,throughput,n_tests\n");
}
void evaluate(
		const std::string implementation_name,
		const std::string input_matrix_name,
		mtk::rsvd_test::rsvd_base& rsvd,
		const unsigned n_tests,
		hipStream_t const hip_stream
		) {
	std::printf("%s,%s,%u,%u,%u,%u,%u,",
			implementation_name.c_str(),
			input_matrix_name.c_str(),
			rsvd.get_m(),
			rsvd.get_n(),
			rsvd.get_k(),
			rsvd.get_p(),
			rsvd.get_n_svdj_iter()
			);
	const auto A_size = rsvd.get_m() * rsvd.get_n();
	const auto S_size = std::min(rsvd.get_m(), rsvd.get_n());
	const auto U_size = rsvd.get_m() * (rsvd.get_k() + rsvd.get_p());
	const auto V_size = rsvd.get_n() * (rsvd.get_k() + rsvd.get_p());

	auto A_ptr = cutf::memory::malloc_async<float>(A_size, hip_stream);
	auto U_ptr = cutf::memory::malloc_async<float>(U_size, hip_stream);
	auto S_ptr = cutf::memory::malloc_async<float>(S_size, hip_stream);
	auto V_ptr = cutf::memory::malloc_async<float>(V_size, hip_stream);

	rsvd.set_input_ptr(A_ptr);
	rsvd.set_output_ptr(U_ptr, S_ptr, V_ptr);

	auto hA_ptr = cutf::memory::malloc_host<float>(A_size);
	rsvd.prepare();

	auto elapsed_time_sum = 0.;
	std::vector<double> residual_list(n_tests);
	std::vector<double> u_orthogonality_list(n_tests);
	std::vector<double> v_orthogonality_list(n_tests);
	for (unsigned i = 0; i < n_tests; i++) {
		// Initialize input matrix
		CUTF_CHECK_ERROR(hipStreamSynchronize(hip_stream));
		mtk::rsvd_test::get_input_matrix(
				hA_ptr, input_matrix_name,
				rsvd.get_m(), rsvd.get_n(),
				i
				);
		cutf::memory::copy_async(A_ptr, hA_ptr, A_size, hip_stream);
		CUTF_CHECK_ERROR(hipStreamSynchronize(hip_stream));

		try {
			hipStreamSynchronize(hip_stream);
			const auto start_clock = std::chrono::system_clock::now();
			rsvd.run();
			hipStreamSynchronize(hip_stream);
			const auto end_clock = std::chrono::system_clock::now();
			const auto elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>(end_clock - start_clock).count() * 1e-6;
			elapsed_time_sum += elapsed_time;
			// Calculate the residual and orthogonality

			residual_list[i] = mtk::mateval::cuda::residual_UxSxVt(
					rsvd.get_m(), rsvd.get_n(), rsvd.get_k(),
					mtk::mateval::col_major, mtk::mateval::col_major, mtk::mateval::col_major,
					U_ptr, rsvd.get_m(),
					S_ptr,
					V_ptr, rsvd.get_n(),
					A_ptr, rsvd.get_m()
					);
			u_orthogonality_list[i] = mtk::mateval::cuda::orthogonality(
					rsvd.get_m(), rsvd.get_k(),
					mtk::mateval::col_major,
					U_ptr, rsvd.get_m()
					);
			v_orthogonality_list[i] = mtk::mateval::cuda::orthogonality(
					rsvd.get_n(), rsvd.get_k(),
					mtk::mateval::col_major,
					V_ptr, rsvd.get_n()
					);
			CUTF_CHECK_ERROR(hipStreamSynchronize(hip_stream));

		} catch (const std::exception& e) {
			std::printf("%s\n", e.what());
		}
	}
	std::printf("%e,%e,%e,",
			mtk::mateval::utils::calc_mean_and_var(residual_list).first,
			mtk::mateval::utils::calc_mean_and_var(u_orthogonality_list).first,
			mtk::mateval::utils::calc_mean_and_var(v_orthogonality_list).first
			);
	std::printf("%e,", elapsed_time_sum / n_tests);

	rsvd.clean();
	cutf::memory::free_async<float>(A_ptr, hip_stream);
	cutf::memory::free_async<float>(U_ptr, hip_stream);
	cutf::memory::free_async<float>(S_ptr, hip_stream);
	cutf::memory::free_async<float>(V_ptr, hip_stream);
	cutf::memory::free_host<float>(hA_ptr);
	std::printf("%u\n", n_tests);
}
} // noname namespace

int main() {
	auto hip_stream  = cutf::stream::get_stream_unique_ptr();
	auto cusolver_handle = cutf::cusolver::dn::get_handle_unique_ptr();
	auto cusolver_params = cutf::cusolver::dn::get_params_unique_ptr();
	CUTF_CHECK_ERROR(hipsolverSetStream(*cusolver_handle.get(), *hip_stream.get()));
	CUTF_CHECK_ERROR(hipsolverDnSetAdvOptions(*cusolver_params.get(), HIPSOLVERDN_GETRF, HIPSOLVER_ALG_0));

	print_csv_header();
	for (unsigned log_m = 5; log_m <= max_log_m; log_m++) {
		for (unsigned log_n = 5; log_n <= max_log_n; log_n++) {
			const auto max_log_k = std::min(log_m, log_n);
			for (unsigned log_k = 4; log_k <= max_log_k - 1; log_k++) {
				const auto m = 1u << log_m;
				const auto n = 1u << log_n;
				const auto k = 1u << log_k;
				const auto p = k;
				if (k + p > std::min(m, n)) {
					break;
				}

				mtk::rsvd_test::rsvd_cusolver rsvd_cusolver(
						*cusolver_handle.get(),
						*cusolver_params.get(),
						m, n, k, p, n_svdj_iter,
						nullptr, m,
						nullptr, m,
						nullptr,
						nullptr, n,
						*hip_stream.get()
						);

				evaluate("cusolver", "latms-" + std::to_string(k), rsvd_cusolver, 10, *hip_stream.get());
			}
		}
	}
}
