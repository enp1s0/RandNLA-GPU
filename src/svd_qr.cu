#include "hip/hip_runtime.h"
#include "svd_base.hpp"
#include <cutf/cusolver.hpp>

std::size_t mtk::rsvd_test::svd_qr::get_working_mem_size_in_byte() {
	int lwork;
	CUTF_CHECK_ERROR(hipsolverDnSgesvd_bufferSize(cusolver_handle, m, n, &lwork));

	const std::size_t r_work = std::min(m, n) - 1;

	work_size = (lwork + r_work) * sizeof(float) + sizeof(int);

	return work_size;
}

void mtk::rsvd_test::svd_qr::run(
		float* const S_ptr,
		float* const U_ptr, const std::size_t ldu,
		float* const V_ptr, const std::size_t ldv,
		float* const input_ptr, const std::size_t ld,
		float* const work_ptr) {

	const std::size_t r_work = std::min(m, n) - 1;
	const std::size_t l_work = work_size - r_work;
	CUTF_CHECK_ERROR(hipsolverDnSgesvd(
				cusolver_handle,
				'S', 'S',
				m, n,
				input_ptr, ld,
				S_ptr,
				U_ptr, ldu,
				V_ptr, ldv,
				work_ptr,
				l_work,
				work_ptr + l_work,
				work_ptr + l_work + 1
				));
}
