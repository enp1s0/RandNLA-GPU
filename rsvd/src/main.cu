#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <chrono>
#include <vector>
#include <rsvd_test.hpp>
#include <input_matrix.hpp>
#include <cutf/memory.hpp>
#include <cutf/cusolver.hpp>
#include <cutf/stream.hpp>
#include <cutf/hiprand.hpp>
#include <mateval/comparison_cuda.hpp>
#include <matfile/matfile.hpp>
#include <fphistogram/fphistogram.hpp>

constexpr unsigned min_log_m = 9;
constexpr unsigned max_log_m = 10;
constexpr unsigned min_log_n = 9;
constexpr unsigned max_log_n = 10;
constexpr unsigned n_tests = 10;
constexpr unsigned n_iter = 0;
constexpr unsigned additional_num_tests_for_time_breakdown = 20;
using svd_t = mtk::rsvd_test::svd_qr;

namespace {
std::vector<std::string> str_split(const std::string str, const char d) {
	std::vector<std::string> strings;
	std::stringstream ss(str);
	std::string s;
	while (getline(ss, s, d)) {
		if (s.length() != 0) {
			strings.push_back(s);
		}
	}
	return strings;
}
void print_csv_header() {
	std::printf("implementation,matrix,m,n,k,p,n_iter,residual,u_orthogonality,v_orthogonality,time,n_tests\n");
}
void evaluate(
		const std::string input_matrix_name,
		mtk::rsvd_test::rsvd_base& rsvd,
		const unsigned n_tests,
		hipStream_t const hip_stream
		) {
	std::printf("%s,%s,%u,%u,%u,%u,%u,",
			rsvd.get_name().c_str(),
			input_matrix_name.c_str(),
			rsvd.get_m(),
			rsvd.get_n(),
			rsvd.get_k(),
			rsvd.get_p(),
			rsvd.get_n_iter()
			);
	const auto A_size = rsvd.get_m() * rsvd.get_n();
	const auto S_size = std::min(rsvd.get_m(), rsvd.get_n());
	const auto U_size = rsvd.get_m() * rsvd.get_k();
	const auto V_size = rsvd.get_n() * rsvd.get_k();

	auto A_ptr = cutf::memory::malloc_async<float>(A_size, hip_stream);
	auto U_ptr = cutf::memory::malloc_async<float>(U_size, hip_stream);
	auto S_ptr = cutf::memory::malloc_async<float>(S_size, hip_stream);
	auto V_ptr = cutf::memory::malloc_async<float>(V_size, hip_stream);

	rsvd.set_input_ptr(A_ptr);
	rsvd.set_output_ptr(U_ptr, S_ptr, V_ptr);

	auto hA_ptr = cutf::memory::malloc_host<float>(A_size);
	rsvd.prepare();

	auto elapsed_time_sum = 0.;
	std::vector<double> residual_list(n_tests);
	std::vector<double> u_orthogonality_list(n_tests);
	std::vector<double> v_orthogonality_list(n_tests);
	for (unsigned i = 0; i < n_tests; i++) {
		// Initialize input matrix
		CUTF_CHECK_ERROR(hipStreamSynchronize(hip_stream));
		mtk::rsvd_test::get_input_matrix(
				hA_ptr, input_matrix_name,
				rsvd.get_m(), rsvd.get_n(),
				i
				);
		cutf::memory::copy_async(A_ptr, hA_ptr, A_size, hip_stream);
		CUTF_CHECK_ERROR(hipStreamSynchronize(hip_stream));

		try {
			rsvd.disable_breakdown_measurement();
			hipStreamSynchronize(hip_stream);
			const auto start_clock = std::chrono::system_clock::now();
			rsvd.run();
			hipStreamSynchronize(hip_stream);
			const auto end_clock = std::chrono::system_clock::now();
			const auto elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>(end_clock - start_clock).count() * 1e-6;
			elapsed_time_sum += elapsed_time;
			// Calculate the residual and orthogonality

			residual_list[i] = mtk::mateval::cuda::residual_UxSxVt(
					rsvd.get_m(), rsvd.get_n(), rsvd.get_k(),
					mtk::mateval::col_major, mtk::mateval::col_major, mtk::mateval::col_major,
					U_ptr, rsvd.get_m(),
					S_ptr,
					V_ptr, rsvd.get_n(),
					hA_ptr, rsvd.get_m()
					);
			u_orthogonality_list[i] = mtk::mateval::cuda::orthogonality(
					rsvd.get_m(), rsvd.get_k(),
					mtk::mateval::col_major,
					U_ptr, rsvd.get_m()
					);
			v_orthogonality_list[i] = mtk::mateval::cuda::orthogonality(
					rsvd.get_n(), rsvd.get_k(),
					mtk::mateval::col_major,
					V_ptr, rsvd.get_n()
					);
			CUTF_CHECK_ERROR(hipStreamSynchronize(hip_stream));
			rsvd.enable_breakdown_measurement();
#ifdef TIME_BREAKDOWN
			for (unsigned i = 0; i < additional_num_tests_for_time_breakdown; i++) {
				rsvd.run();
			}
#endif

		} catch (const std::exception& e) {
			std::printf("%s\n", e.what());
		}
	}
	std::printf("%e,%e,%e,",
			mtk::mateval::utils::calc_mean_and_var(residual_list).first,
			mtk::mateval::utils::calc_mean_and_var(u_orthogonality_list).first,
			mtk::mateval::utils::calc_mean_and_var(v_orthogonality_list).first
			);
	std::printf("%e,", elapsed_time_sum / n_tests);

	rsvd.clean();
	cutf::memory::free_async<float>(A_ptr, hip_stream);
	cutf::memory::free_async<float>(U_ptr, hip_stream);
	cutf::memory::free_async<float>(S_ptr, hip_stream);
	cutf::memory::free_async<float>(V_ptr, hip_stream);
	cutf::memory::free_host<float>(hA_ptr);
	std::printf("%u\n", n_tests);
}

void breakdown_eval() {
	auto hip_stream  = cutf::stream::get_stream_unique_ptr();
	auto cusolver_handle = cutf::cusolver::dn::get_handle_unique_ptr();
	auto cusolver_params = cutf::cusolver::dn::get_params_unique_ptr();
	auto cublas_handle = cutf::cublas::get_cublas_unique_ptr();
	CUTF_CHECK_ERROR(hipsolverSetStream(*cusolver_handle.get(), *hip_stream.get()));
	CUTF_CHECK_ERROR(hipsolverDnSetAdvOptions(*cusolver_params.get(), HIPSOLVERDN_GETRF, HIPSOLVER_ALG_0));
	CUTF_CHECK_ERROR(hipblasSetStream(*cublas_handle.get(), *hip_stream.get()));

	mtk::shgemm::shgemmHandle_t shgemm_handle;
	mtk::shgemm::create(shgemm_handle);
	mtk::shgemm::set_cuda_stream(shgemm_handle, *hip_stream.get());

	print_csv_header();
	for (unsigned log_m = min_log_n; log_m <= max_log_m; log_m += 2) {
		//for (unsigned log_n = min_log_n; log_n <= max_log_n; log_n++) {
		{
			const auto log_n = log_m;
			const auto max_log_k = std::min(log_m, log_n);
			for (unsigned log_k = 6; log_k <= max_log_k - 4; log_k++) {
				const auto m = 1u << log_m;
				const auto n = 1u << log_n;
				const auto k = 1u << log_k;
				const auto decomp_k = k;
				const auto p = decomp_k / 10;
				if (decomp_k + p > std::min(m, n)) {
					break;
				}

				const std::string matrix_name = "latms-" + std::to_string(k);
				
				svd_t svd(*cusolver_handle.get());
				{
					mtk::rsvd_test::random_projection_fp32 rand_proj_fp32(*cublas_handle.get());
					mtk::rsvd_test::rsvd_selfmade rsvd_selfmade(
							*cublas_handle.get(),
							*cusolver_handle.get(),
							*cusolver_params.get(),
							m, n, decomp_k, p, n_iter,
							nullptr, m,
							nullptr, m,
							nullptr,
							nullptr, n,
							*hip_stream.get(),
							svd,
							rand_proj_fp32
							);
					evaluate(matrix_name, rsvd_selfmade, n_tests, *hip_stream.get());
					std::printf("# START human time-breakdown-%s-%u-%u-%u-%u-%s\n", matrix_name.c_str(), m, n, decomp_k, p, rand_proj_fp32.get_name().c_str());
					rsvd_selfmade.print_time_breakdown();
					std::printf("# END human\n");
					std::printf("# START csv time-breakdown-%s-%u-%u-%u-%u-%s\n", matrix_name.c_str(), m, n, decomp_k, p, rand_proj_fp32.get_name().c_str());
					rsvd_selfmade.print_time_breakdown(true);
					std::printf("# END csv\n");
				}
				{
					mtk::rsvd_test::random_projection_tf32 rand_proj_tf32(*cublas_handle.get());
					mtk::rsvd_test::rsvd_selfmade rsvd_selfmade(
							*cublas_handle.get(),
							*cusolver_handle.get(),
							*cusolver_params.get(),
							m, n, decomp_k, p, n_iter,
							nullptr, m,
							nullptr, m,
							nullptr,
							nullptr, n,
							*hip_stream.get(),
							svd,
							rand_proj_tf32
							);
					evaluate(matrix_name, rsvd_selfmade, n_tests, *hip_stream.get());
					std::printf("# START human time-breakdown-%s-%u-%u-%u-%u-%s\n", matrix_name.c_str(), m, n, decomp_k, p, rand_proj_tf32.get_name().c_str());
					rsvd_selfmade.print_time_breakdown();
					std::printf("# END human\n");
					std::printf("# START csv time-breakdown-%s-%u-%u-%u-%u-%s\n", matrix_name.c_str(), m, n, decomp_k, p, rand_proj_tf32.get_name().c_str());
					rsvd_selfmade.print_time_breakdown(true);
					std::printf("# END csv\n");
				}
				{
					mtk::rsvd_test::random_projection_shgemm rand_proj_shgemm(shgemm_handle, mtk::shgemm::tf32);
					mtk::rsvd_test::rsvd_selfmade rsvd_selfmade(
							*cublas_handle.get(),
							*cusolver_handle.get(),
							*cusolver_params.get(),
							m, n, decomp_k, p, n_iter,
							nullptr, m,
							nullptr, m,
							nullptr,
							nullptr, n,
							*hip_stream.get(),
							svd,
							rand_proj_shgemm
							);
					evaluate(matrix_name, rsvd_selfmade, n_tests, *hip_stream.get());
					std::printf("# START human time-breakdown-%s-%u-%u-%u-%u-%s\n", matrix_name.c_str(), m, n, decomp_k, p, rand_proj_shgemm.get_name().c_str());
					rsvd_selfmade.print_time_breakdown();
					std::printf("# END human\n");
					std::printf("# START csv time-breakdown-%s-%u-%u-%u-%u-%s\n", matrix_name.c_str(), m, n, decomp_k, p, rand_proj_shgemm.get_name().c_str());
					rsvd_selfmade.print_time_breakdown(true);
					std::printf("# END csv\n");
				}
				{
					mtk::rsvd_test::random_projection_shgemm rand_proj_shgemm(shgemm_handle, mtk::shgemm::fp16);
					mtk::rsvd_test::rsvd_selfmade rsvd_selfmade(
							*cublas_handle.get(),
							*cusolver_handle.get(),
							*cusolver_params.get(),
							m, n, decomp_k, p, n_iter,
							nullptr, m,
							nullptr, m,
							nullptr,
							nullptr, n,
							*hip_stream.get(),
							svd,
							rand_proj_shgemm
							);
					evaluate(matrix_name, rsvd_selfmade, n_tests, *hip_stream.get());
					std::printf("# START human time-breakdown-%s-%u-%u-%u-%u-%s\n", matrix_name.c_str(), m, n, decomp_k, p, rand_proj_shgemm.get_name().c_str());
					rsvd_selfmade.print_time_breakdown();
					std::printf("# END human\n");
					std::printf("# START csv time-breakdown-%s-%u-%u-%u-%u-%s\n", matrix_name.c_str(), m, n, decomp_k, p, rand_proj_shgemm.get_name().c_str());
					rsvd_selfmade.print_time_breakdown(true);
					std::printf("# END csv\n");
				}
			}
		}
	}
	mtk::shgemm::destroy(shgemm_handle);
}

void accuracy_test() {
	auto hip_stream  = cutf::stream::get_stream_unique_ptr();
	auto cusolver_handle = cutf::cusolver::dn::get_handle_unique_ptr();
	auto cusolver_params = cutf::cusolver::dn::get_params_unique_ptr();
	auto cublas_handle = cutf::cublas::get_cublas_unique_ptr();
	CUTF_CHECK_ERROR(hipsolverSetStream(*cusolver_handle.get(), *hip_stream.get()));
	CUTF_CHECK_ERROR(hipsolverDnSetAdvOptions(*cusolver_params.get(), HIPSOLVERDN_GETRF, HIPSOLVER_ALG_0));
	CUTF_CHECK_ERROR(hipblasSetStream(*cublas_handle.get(), *hip_stream.get()));

	mtk::shgemm::shgemmHandle_t shgemm_handle;
	mtk::shgemm::create(shgemm_handle);
	mtk::shgemm::set_cuda_stream(shgemm_handle, *hip_stream.get());

	std::vector<std::string> matrix_list = {"latms", "latms_sigmoid"};

	print_csv_header();
	for (const auto& matrix : matrix_list) {
	for (unsigned log_m = min_log_n; log_m <= max_log_m; log_m++) {
		//for (unsigned log_n = min_log_n; log_n <= max_log_n; log_n++) {
		{
			const auto log_n = log_m;
			const auto max_log_k = std::min(log_m, log_n);
			for (unsigned rank_index = 0; rank_index < 3; rank_index++) {
				const auto m = 1u << log_m;
				const auto n = 1u << log_n;

				unsigned rank = 0;
				unsigned k = 0;
				if (matrix == "latms") {
					rank = std::min(m, n) / 32;
					switch(rank_index) {
						case 0: k = rank * 999 / 1000; break;
						case 1: k = rank * 1000 / 1000; break;
						case 2: k = rank * 1001 / 1000; break;
						default: break;
					}
				} else if (matrix == "latms_sigmoid") {
					rank = std::min(m, n) / 32;
					switch(rank_index) {
						case 0: k = rank * 15 / 10; break;
						case 1: k = rank * 2; break;
						case 2: k = rank * 4; break;
						default: break;
					}
				}
				const auto decomp_k = k;
				const auto p = 32;
				if (decomp_k + p > std::min(m, n)) {
					break;
				}

				const std::string matrix_name = matrix + "-" + std::to_string(rank);
				
				svd_t svd(*cusolver_handle.get());

#if defined(RUN_REFERENCE_FUNCTIONS) && !defined(TIME_BREAKDOWN)
				mtk::rsvd_test::rsvd_cusolver rsvd_cusolver(
						*cusolver_handle.get(),
						*cusolver_params.get(),
						m, n, decomp_k, p, n_iter,
						nullptr, m,
						nullptr, m,
						nullptr,
						nullptr, n,
						*hip_stream.get()
						);
				evaluate(matrix_name, rsvd_cusolver, n_tests, *hip_stream.get());
#endif

				{
					mtk::rsvd_test::random_projection_fp32 rand_proj_fp32(*cublas_handle.get());
					mtk::rsvd_test::rsvd_selfmade rsvd_selfmade(
							*cublas_handle.get(),
							*cusolver_handle.get(),
							*cusolver_params.get(),
							m, n, decomp_k, p, n_iter,
							nullptr, m,
							nullptr, m,
							nullptr,
							nullptr, n,
							*hip_stream.get(),
							svd,
							rand_proj_fp32
							);
					evaluate(matrix_name, rsvd_selfmade, n_tests, *hip_stream.get());
				}
				{
					mtk::rsvd_test::random_projection_tf32 rand_proj_tf32(*cublas_handle.get());
					mtk::rsvd_test::rsvd_selfmade rsvd_selfmade(
							*cublas_handle.get(),
							*cusolver_handle.get(),
							*cusolver_params.get(),
							m, n, decomp_k, p, n_iter,
							nullptr, m,
							nullptr, m,
							nullptr,
							nullptr, n,
							*hip_stream.get(),
							svd,
							rand_proj_tf32
							);
					evaluate(matrix_name, rsvd_selfmade, n_tests, *hip_stream.get());
				}
				{
					mtk::rsvd_test::random_projection_shgemm rand_proj_shgemm(shgemm_handle, mtk::shgemm::tf32);
					mtk::rsvd_test::rsvd_selfmade rsvd_selfmade(
							*cublas_handle.get(),
							*cusolver_handle.get(),
							*cusolver_params.get(),
							m, n, decomp_k, p, n_iter,
							nullptr, m,
							nullptr, m,
							nullptr,
							nullptr, n,
							*hip_stream.get(),
							svd,
							rand_proj_shgemm
							);
					evaluate(matrix_name, rsvd_selfmade, n_tests, *hip_stream.get());
				}
				{
					mtk::rsvd_test::random_projection_shgemm rand_proj_shgemm(shgemm_handle, mtk::shgemm::fp16);
					mtk::rsvd_test::rsvd_selfmade rsvd_selfmade(
							*cublas_handle.get(),
							*cusolver_handle.get(),
							*cusolver_params.get(),
							m, n, decomp_k, p, n_iter,
							nullptr, m,
							nullptr, m,
							nullptr,
							nullptr, n,
							*hip_stream.get(),
							svd,
							rand_proj_shgemm
							);
					evaluate(matrix_name, rsvd_selfmade, n_tests, *hip_stream.get());
				}

#if defined(RUN_REFERENCE_FUNCTIONS) && !defined(TIME_BREAKDOWN)
				mtk::rsvd_test::svdj_cusolver svdj_cusolver(
						*cusolver_handle.get(),
						m, n, decomp_k, p, n_iter,
						nullptr, m,
						nullptr, m,
						nullptr,
						nullptr, n,
						*hip_stream.get()
						);
				evaluate(matrix_name, svdj_cusolver, n_tests, *hip_stream.get());
#endif
			}
		}
	}
	}
	mtk::shgemm::destroy(shgemm_handle);
}

void watermark_core(
		mtk::rsvd_test::rsvd_base& rsvd,
		const std::string output_dir,
		const std::string base_name,
		const float* const u_ptr,
		const float* const s_ptr,
		const float* const v_ptr
		) {
	rsvd.prepare();
	rsvd.run();
	hipDeviceSynchronize();

	const auto m = rsvd.get_m();
	const auto n = rsvd.get_n();
	const auto decomp_k = rsvd.get_k();

	mtk::matfile::save_dense(decomp_k, 1, s_ptr, decomp_k, output_dir + "/" + base_name + "." + rsvd.get_name() + ".s.matrix");
	mtk::matfile::save_dense(m, decomp_k, u_ptr, m,        output_dir + "/" + base_name + "." + rsvd.get_name() + ".u.matrix");
	mtk::matfile::save_dense(n, decomp_k, v_ptr, n,        output_dir + "/" + base_name + "." + rsvd.get_name() + ".v.matrix");

	std::printf("[%s] Largest sv = %e\n", rsvd.get_name().c_str(), s_ptr[0]);

	rsvd.clean();
}

void watermark(
		const std::string list_file_name,
		const std::string output_dir,
		const std::size_t max_image_width,
		const std::size_t max_image_height
		) {
	auto hip_stream  = cutf::stream::get_stream_unique_ptr();
	auto cusolver_handle = cutf::cusolver::dn::get_handle_unique_ptr();
	auto cusolver_params = cutf::cusolver::dn::get_params_unique_ptr();
	auto cublas_handle = cutf::cublas::get_cublas_unique_ptr();
	CUTF_CHECK_ERROR(hipsolverSetStream(*cusolver_handle.get(), *hip_stream.get()));
	CUTF_CHECK_ERROR(hipsolverDnSetAdvOptions(*cusolver_params.get(), HIPSOLVERDN_GETRF, HIPSOLVER_ALG_0));
	CUTF_CHECK_ERROR(hipblasSetStream(*cublas_handle.get(), *hip_stream.get()));

	mtk::shgemm::shgemmHandle_t shgemm_handle;
	mtk::shgemm::create(shgemm_handle);
	mtk::shgemm::set_cuda_stream(shgemm_handle, *hip_stream.get());


	auto image_matrix_uptr = cutf::memory::get_host_unique_ptr<float>(max_image_height * max_image_width * 3);

	const auto max_rank = std::min(max_image_height, max_image_width * 3);

	auto s_uptr = cutf::memory::get_host_unique_ptr<float>(max_rank);
	auto u_uptr = cutf::memory::get_host_unique_ptr<float>(max_image_height * max_rank);
	auto v_uptr = cutf::memory::get_host_unique_ptr<float>(max_image_width * 3 * max_rank);

	svd_t svd(*cusolver_handle.get());

	std::ifstream ifs(list_file_name);
	std::string file_name;
	while (std::getline(ifs, file_name)) {
		std::size_t w, h;
		mtk::matfile::load_size(h, w, file_name);
		std::printf("file_name = %s\n", file_name.c_str());
		std::printf("image_matrix = (%lu x %lu)\n", w, h);
		std::fflush(stdout);

		const auto tmp_str_list = str_split(file_name, '/');
		for (const auto& s : tmp_str_list) std::printf("%s ", s.c_str());
		std::printf("\n");
		const auto tmp_str_list_2 = str_split(tmp_str_list[tmp_str_list.size() - 1], '.');
		auto base_name = tmp_str_list_2[0];
		for (unsigned i = 0; i < tmp_str_list_2.size() - 2; i++) {
			base_name += "." + tmp_str_list_2[i];
		}

		std::printf("base_name = %s\n", base_name.c_str());
		std::fflush(stdout);


		const auto m = h;
		const auto n = w;
		const auto p = 100lu;
		const auto decomp_k = std::stoul(tmp_str_list[tmp_str_list.size() - 2]);

		std::printf("input=(%lu, %lu), k = %lu, p = %lu\n", m, n, decomp_k, p);
		std::fflush(stdout);

		mtk::matfile::load_dense(image_matrix_uptr.get(), h, file_name);
		mtk::fphistogram::print_histogram<float, mtk::fphistogram::mode_log10>(image_matrix_uptr.get(), max_image_height * max_image_width);
		printf("(2,1) = [[%e], [%e]]\n", image_matrix_uptr.get()[0], image_matrix_uptr.get()[1]);

		// RSVD
		{
			mtk::rsvd_test::random_projection_fp32 rand_proj(*cublas_handle.get());
			mtk::rsvd_test::rsvd_selfmade rsvd(
					*cublas_handle.get(),
					*cusolver_handle.get(),
					*cusolver_params.get(),
					m, n, decomp_k, p, n_iter,
					image_matrix_uptr.get(), m,
					u_uptr.get(), m,
					s_uptr.get(),
					v_uptr.get(), n,
					*hip_stream.get(),
					svd,
					rand_proj
					);

			// load
			watermark_core(rsvd, output_dir, base_name, u_uptr.get(), s_uptr.get(), v_uptr.get());
		}
		{
			mtk::rsvd_test::random_projection_tf32 rand_proj(*cublas_handle.get());
			mtk::rsvd_test::rsvd_selfmade rsvd(
					*cublas_handle.get(),
					*cusolver_handle.get(),
					*cusolver_params.get(),
					m, n, decomp_k, p, n_iter,
					image_matrix_uptr.get(), m,
					u_uptr.get(), m,
					s_uptr.get(),
					v_uptr.get(), n,
					*hip_stream.get(),
					svd,
					rand_proj
					);

			// load
			watermark_core(rsvd, output_dir, base_name, u_uptr.get(), s_uptr.get(), v_uptr.get());
		}
		{
			mtk::rsvd_test::random_projection_shgemm rand_proj(shgemm_handle, mtk::shgemm::fp16);
			mtk::rsvd_test::rsvd_selfmade rsvd(
					*cublas_handle.get(),
					*cusolver_handle.get(),
					*cusolver_params.get(),
					m, n, decomp_k, p, n_iter,
					image_matrix_uptr.get(), m,
					u_uptr.get(), m,
					s_uptr.get(),
					v_uptr.get(), n,
					*hip_stream.get(),
					svd,
					rand_proj
					);

			// load
			watermark_core(rsvd, output_dir, base_name, u_uptr.get(), s_uptr.get(), v_uptr.get());
		}
		{
			mtk::rsvd_test::random_projection_shgemm rand_proj(shgemm_handle, mtk::shgemm::tf32);
			mtk::rsvd_test::rsvd_selfmade rsvd(
					*cublas_handle.get(),
					*cusolver_handle.get(),
					*cusolver_params.get(),
					m, n, decomp_k, p, n_iter,
					image_matrix_uptr.get(), m,
					u_uptr.get(), m,
					s_uptr.get(),
					v_uptr.get(), n,
					*hip_stream.get(),
					svd,
					rand_proj
					);

			// load
			watermark_core(rsvd, output_dir, base_name, u_uptr.get(), s_uptr.get(), v_uptr.get());
		}
		{
			mtk::rsvd_test::rsvd_cusolver rsvd(
					*cusolver_handle.get(),
					*cusolver_params.get(),
					m, n, decomp_k, p, n_iter,
					image_matrix_uptr.get(), m,
					u_uptr.get(), m,
					s_uptr.get(),
					v_uptr.get(), n,
					*hip_stream.get()
					);

			// load
			watermark_core(rsvd, output_dir, base_name, u_uptr.get(), s_uptr.get(), v_uptr.get());
		}
		{
			mtk::rsvd_test::svdj_cusolver rsvd(
					*cusolver_handle.get(),
					m, n, decomp_k, p, n_iter,
					image_matrix_uptr.get(), m,
					u_uptr.get(), m,
					s_uptr.get(),
					v_uptr.get(), n,
					*hip_stream.get()
					);

			// load
			watermark_core(rsvd, output_dir, base_name, u_uptr.get(), s_uptr.get(), v_uptr.get());
		}
	}
}
} // noname namespace

int main(int argc, char** argv) {
	if (argc == 4 && std::string(argv[1]) == "watermark") {
		watermark(argv[2], argv[3], 4032, 4032);
	} else if (argc == 2 && std::string(argv[1]) == "breakdown") {
		breakdown_eval();
	} else {
		accuracy_test();
	}
}
